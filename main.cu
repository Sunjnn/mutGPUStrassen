#include <hip/hip_runtime.h>
#include <hipblas.h>

#include <stdio.h>
#include <stdlib.h>
#include <omp.h>

#include "main.cuh"
#include "cudaUti.cuh"
#include "gemmStrassen.cuh"
#include "matrixUti.hxx"
#include "blockMatrix.hxx"


int main() {
    float *A = (float*)malloc(sizeof(float) * M * K);
    float *B = (float*)malloc(sizeof(float) * K * N);
    float *C = (float*)malloc(sizeof(float) * M * N);
    float *CTest = (float*)malloc(sizeof(float) * M * N);

    initMatrix(A, M, K);
    initMatrix(B, K, N);

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    float time;

    hipblasHandle_t handle;
    hipblasCreate(&handle);
    gemmcublas(CTest, A, B, M, K, N, handle);

    memset(CTest, 0, sizeof(float) * M * N);

    hipDeviceSynchronize();
    hipEventRecord(start, 0);
    gemmstrassenNOomp(CTest, A, B, M, K, N, handle);
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&time, start, stop);
    printf("gemmstrassenNOomp time: %f ms\n", time);
    // test(C, CTest, M, N);

    memset(C, 0, sizeof(float) * M * N);
    time = 0.0f;

    hipDeviceSynchronize();
    hipEventRecord(start, 0);
    gemmstrassen(C, A, B, M, K, N, M, K, N);
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&time, start, stop);
    printf("gemmstrassen time: %f ms\n", time);
    test(C, CTest, M, N);

    return 0;
}
