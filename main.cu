#include <hip/hip_runtime.h>
#include <hipblas.h>

#include <stdio.h>
#include <stdlib.h>
#include <omp.h>
#include <functional>
#include <vector>

#include "main.cuh"
#include "cudaUti.cuh"
#include "gemmStrassen.cuh"
#include "matrixUti.hxx"
#include "blockMatrix.hxx"
#include "threadPool.cuh"


int main() {
    float *A = (float*)malloc(sizeof(float) * M * K);
    float *B = (float*)malloc(sizeof(float) * K * N);
    float *C = (float*)malloc(sizeof(float) * M * N);
    float *CTest = (float*)malloc(sizeof(float) * M * N);

    initMatrix(A, M, K);
    initMatrix(B, K, N);

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    float time;

    hipblasHandle_t handle;
    hipblasCreate(&handle);
    hipStream_t stream;
    hipStreamCreate(&stream);
    gemmcublas(CTest, A, B, M, K, N, handle);

    // memset(CTest, 0, sizeof(float) * M * N);

    // hipDeviceSynchronize();
    // hipEventRecord(start, 0);
    // gemmcublas(CTest, A, B, M, K, N, handle, stream);
    // hipEventRecord(stop, 0);
    // hipEventSynchronize(stop);
    // hipEventElapsedTime(&time, start, stop);
    // printf("gemmcublas time: %f ms\n", time);
    // // test(C, CTest, M, N);

    // hipDeviceSynchronize();
    // hipEventRecord(start, 0);
    // gemmstrassenNOomp(CTest, A, B, M, K, N, handle);
    // hipEventRecord(stop, 0);
    // hipEventSynchronize(stop);
    // hipEventElapsedTime(&time, start, stop);
    // printf("gemmstrassenNOomp time: %f ms\n", time);
    // // test(C, CTest, M, N);

    // memset(C, 0, sizeof(float) * M * N);
    // time = 0.0f;

    // hipDeviceSynchronize();
    // hipEventRecord(start, 0);
    // gemmstrassen(C, A, B, M, K, N, M, K, M);
    // hipEventRecord(stop, 0);
    // hipEventSynchronize(stop);
    // hipEventElapsedTime(&time, start, stop);
    // printf("gemmstrassen time: %f ms\n", time);
    // test(C, CTest, M, N);

    // memset(C, 0, sizeof(float) * M * N);
    // time = 0.0f;

    // hipDeviceSynchronize();
    // hipEventRecord(start, 0);
    // gemmstrassen_v3(C, M, A, M, B, K, M);
    // hipEventRecord(stop, 0);
    // hipEventSynchronize(stop);
    // hipEventElapsedTime(&time, start, stop);
    // printf("gemmstrassen_v3 time: %f ms\n", time);
    // test(C, CTest, M, N);

    memset(C, 0, sizeof(float) * M * N);
    time = 0.0f;

    hipDeviceSynchronize();
    hipEventRecord(start, 0);
    std::vector<int> GPUs;
    GPUs.push_back(0);
    threadPoolConfig config(C, A, B, M, K, N, BLOCK_M, BLOCK_K, BLOCK_N, GPUs);
    threadCPU(&config);
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&time, start, stop);
    printf("gemmstrassen threadPool time: %f ms\n", time);
    test(C, CTest, M, N);

    return 0;
}
