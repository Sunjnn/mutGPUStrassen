#include <hip/hip_runtime.h>
#include <hipblas.h>

#include <vector>
#include <thread>
#include <functional>
#include <stdio.h>

#include "gemmStrassen.cuh"
#include "threadPool.cuh"
#include "cudaUti.cuh"
#include "matrixUti.hxx"

threadPoolConfig::threadPoolConfig() {}

threadPoolConfig::threadPoolConfig(float* C, float* A, float* B, int M, int K, int N, int blockM, int blockK, int blockN, std::vector<int> GPUsArray): m(M), k(K), n(N), blockM(blockM), blockK(blockK), blockN(blockN) {
    bmatA = new blockMatrix(A, M, K, M, blockM, blockK);
    bmatB = new blockMatrix(B, K, N, K, blockK, blockN);
    bmatC = new blockMatrix(C, M, N, M, blockM, blockN);

    deviceCount = getdevicecount();
    for (int i = 0; i < GPUsArray.size(); ++i) {
        if (GPUsArray[i] >= deviceCount) {
            printf("%d GPU does not exist.\n", GPUsArray[i]);
        }

        memMiBs.push_back(getdeviceprop(GPUsArray[i]));
        GPUs.push_back(GPUsArray[i]);
    }
    deviceCount = GPUsArray.size();

    for (int i = 0; i < bmatC->dimM; ++i) {
        for (int j = 0; j < bmatC->dimN; ++j) {
            //int task[2] = { i, j };
            int *task = new int[2];
            task[0] = i;
            task[1] = j;
            tasks.push_back(task);
        }
    }
}

void threadGPUSub(threadPoolConfig *config, int start, int stop) {
    float *CTmp = (float*)malloc(sizeof(float) * config->blockM * config->blockN);
    hipStream_t *streamArray = (hipStream_t*)malloc(sizeof(hipStream_t) * 3);
    hipblasHandle_t *handleArray = (hipblasHandle_t*)malloc(sizeof(hipblasHandle_t) * 3);
    for (int i = 0; i < 3; ++i) {
        CHECKCUDA(hipStreamCreate(streamArray + i));
        CHECKCUBLAS(hipblasCreate(handleArray + i));
        CHECKCUBLAS(hipblasSetStream(handleArray[i], streamArray[i]));
    }

    float *T1 = nullptr;
    CHECKCUDA(hipMallocAsync(&T1, sizeof(float) * config->blockM * config->blockM, streamArray[0]));
    float *T2 = nullptr;
    CHECKCUDA(hipMallocAsync(&T2, sizeof(float) * config->blockM * config->blockM, streamArray[1]));

    float *d_A = nullptr;
    CHECKCUDA(hipMallocAsync(&d_A, sizeof(float) * config->blockM * config->blockM, streamArray[0]))

    float *d_B = nullptr;
    CHECKCUDA(hipMallocAsync(&d_B, sizeof(float) * config->blockM * config->blockM, streamArray[1]));

    float *d_C = nullptr;
    CHECKCUDA(hipMallocAsync(&d_C, sizeof(float) * config->blockM * config->blockM, streamArray[2]));

    for (int index = start; index < stop; ++index) {
        int i = config->tasks[index][0];
        int j = config->tasks[index][1];
        for (int k = 0; k < config->bmatA->dimN; ++k) {
            // gemmstrassen(config->bmatC->getBlockMatrix(i, j), config->bmatA->getBlockMatrix(i, k), config->bmatB->getBlockMatrix(k, j), config->blockM, config->blockK, config->blockN, config->m, config->k, config->m);
            // gemmstrassen_v2(config->bmatC->getBlockMatrix(i, j), config->m, config->bmatA->getBlockMatrix(i, k), config->m, config->bmatB->getBlockMatrix(k, j), config->k, config->blockM, config->blockK, config->blockN);

            gemmstrassen_v3(CTmp, config->blockM, config->bmatA->getBlockMatrix(i, k), config->m, config->bmatB->getBlockMatrix(k, j), config->k, config->blockM, streamArray, handleArray, T1, T2, d_A, d_B, d_C);
            matrixAdd(config->bmatC->getBlockMatrix(i, j), config->bmatC->getBlockMatrix(i, j), CTmp, config->blockM, config->blockN, config->m, config->m, config->blockM);
        }
    }
    CHECKCUDA(hipFreeAsync(T1, streamArray[0]));
    CHECKCUDA(hipFreeAsync(T2, streamArray[1]));
    CHECKCUDA(hipFreeAsync(d_A, streamArray[0]));
    CHECKCUDA(hipFreeAsync(d_B, streamArray[1]));
    CHECKCUDA(hipFreeAsync(d_C, streamArray[2]));
    free(CTmp);
    for (int i = 0; i < 3; ++i) {
        CHECKCUDA(hipStreamDestroy(streamArray[i]));
        CHECKCUBLAS(hipblasDestroy(handleArray[i]));
    }
    free(streamArray);
    free(handleArray);
}

void threadGPUMas(threadPoolConfig *config, int dev, int threadNum, int start, int stop) {
    CHECKCUDA(hipSetDevice(dev));

    int iterLen = stop - start;
    int iterNumThread = iterLen / threadNum;
    std::thread **threadArray = (std::thread**)malloc(sizeof(std::thread*) * threadNum);
    // std::vector<std::thread> threadArray;

    int i = 0;
    for (i = 0; i < threadNum - 1; ++i) {
        // threadArray.push_back(std::move(std::thread(threadGPUSub, C, A, B, iterBegin + i * iterNumThread, iterBegin + (i + 1) * iterNumThread)));
        threadArray[i] = new std::thread(threadGPUSub, config, start + i * iterNumThread, start + (i + 1) * iterNumThread);
        
    }
    // threadArray.push_back(std::move(std::thread(threadGPUSub, C, A, B, iterBegin + i * iterNumThread, iterEnd)));
    threadArray[i] = new std::thread(threadGPUSub, config, start + i * iterNumThread, stop);
    

    for (i = 0; i < threadNum; ++i) {
        threadArray[i]->join();
    }
}

void threadCPU(threadPoolConfig *config) {
    float mem = (config->blockM * config->blockK + config->blockK * config->blockN + config->blockM * config->blockN) / 1048576.0f * 7.0f;
    int taskLenGPU = (config->tasks.end() - config->tasks.begin()) / config->deviceCount;
    std::thread **threadArray = (std::thread**)malloc(sizeof(std::thread*) * config->deviceCount);
    // std::vector<std::thread> threadArray;

    int dev = 0;
    int num_thread;
    for (dev = 0; dev < config->deviceCount - 1; ++dev)
    {
        num_thread = config->memMiBs[dev] / mem;
        if (num_thread > 3) num_thread = 3;

        // threadArray.push_back(std::move(std::thread(threadGPUMas, bmatC, bmatA, bmatB, num_thread, tasks.begin() + dev * taskLenGPU, tasks.begin() + (dev + 1) * taskLenGPU)));
        threadArray[dev] = new std::thread(threadGPUMas, config, config->GPUs[dev], num_thread, dev * taskLenGPU, (dev + 1) * taskLenGPU);
    }
    num_thread = config->memMiBs[dev] / mem;
    if (num_thread > 8) num_thread = 8;
    // threadArray.push_back(std::move(std::thread(threadGPUMas, bmatC, bmatA, bmatB, tasks.begin() + dev * taskLenGPU, tasks.end())));
    printf("ready for threadGPUMas %d\n", dev);
    threadArray[dev] = new std::thread(threadGPUMas, config, config->GPUs[dev], num_thread, dev * taskLenGPU, config->tasks.size());

    for (dev = 0; dev < config->deviceCount; ++dev) {
        threadArray[dev]->join();
    }
}