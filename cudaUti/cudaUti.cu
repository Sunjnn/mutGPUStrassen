#include <hip/hip_runtime.h>
#include <hipblas.h>

#include <stdio.h>

#include "cudaUti.cuh"


// basic routine. compute C = AB.
// C, A and B must be consecutive.
// could set the cuda stream.
void gemmcublas(float *C, float *A, float *B, int M, int K, int N, hipblasHandle_t handle, hipStream_t stream) {
    hipblasStatus_t hipblasStatus_t = HIPBLAS_STATUS_SUCCESS;
    hipblasStatus_t = hipblasSetStream(handle, stream);
    CHECKCUBLAS(hipblasStatus_t);

    //void *workspace{ nullptr };
    //CHECKCUDA(hipMallocAsync(&workspace, 1024 * 1024, stream));
    //CHECKCUBLAS(hipblasSetWorkspace(handle, workspace, 1024 * 1024));

    hipError_t cudaStatus;
    float *d_A{nullptr}, *d_B{nullptr}, *d_C{nullptr};
    cudaStatus = hipMallocAsync(&d_A, sizeof(float) * M * K, stream);
    CHECKCUDA(cudaStatus);
    cudaStatus = hipMallocAsync(&d_B, sizeof(float) * K * N, stream);
    CHECKCUDA(cudaStatus);
    cudaStatus = hipMallocAsync(&d_C, sizeof(float) * M * N, stream);
    CHECKCUDA(cudaStatus);

    hipblasStatus_t = hipblasSetMatrixAsync(M, K, sizeof(float), A, M, d_A, M, stream);
    CHECKCUBLAS(hipblasStatus_t);
    hipblasStatus_t = hipblasSetMatrixAsync(K, N, sizeof(float), B, K, d_B, K, stream);
    CHECKCUBLAS(hipblasStatus_t);

    float alpha = 1.0f, beta = 0.0f;

    hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, M, N, N, &alpha, d_A, M, d_B, K, &beta, d_C, M);
    hipblasStatus_t = hipblasGetMatrixAsync(M, N, sizeof(float), d_C, M, C, M, stream);
    CHECKCUBLAS(hipblasStatus_t);

    cudaStatus = hipFreeAsync(d_A, stream);
    CHECKCUDA(cudaStatus);
    cudaStatus = hipFreeAsync(d_B, stream);
    CHECKCUDA(cudaStatus);
    cudaStatus = hipFreeAsync(d_C, stream);
    CHECKCUDA(cudaStatus);
}

// basic routine. compute C = AB.
// C, A and B must be consecutive.
void gemmcublas(float *C, float *A, float *B, int M, int K, int N, hipblasHandle_t handle) {
    float *d_A{nullptr}, *d_B{nullptr}, *d_C{nullptr};
    hipMalloc(&d_A, sizeof(float) * M * K);
    hipMalloc(&d_B, sizeof(float) * K * N);
    hipMalloc(&d_C, sizeof(float) * M * N);

    hipblasSetMatrix(M, K, sizeof(float), A, M, d_A, M);
    hipblasSetMatrix(K, N, sizeof(float), B, K, d_B, K);

    float alpha = 1.0f, beta = 0.0f;

    hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, M, N, K, &alpha, d_A, M, d_B, K, &beta, d_C, M);
    hipblasGetMatrix(N, N, sizeof(float), d_C, M, C, M);

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
}

// get the number of system.
int getdevicecount() {
    int deviceCount = 0;
    CHECKCUDA(hipGetDeviceCount(&deviceCount));
    return deviceCount;
}

// get properties of each GPU: memory.
float *getdeviceprop(int deviceCount) {
    if (deviceCount == 0) {
        printf("There are no avilable device(s) that support CUDA\n");
        exit(1);
    }
    else {
        printf("Delete %d CUDA Capable device(s)\n", deviceCount);
    }

    float* memMiBs = (float*)malloc(sizeof(float) * deviceCount);

    for (int dev = 0; dev < deviceCount; ++dev) {
        //CHECKCUDA(hipSetDevice(dev));
        hipDeviceProp_t deviceProp;
        CHECKCUDA(hipGetDeviceProperties(&deviceProp, dev));
        //float memMiB = deviceProp.totalGlobalMem / 1048576.0f;
        memMiBs[dev] = deviceProp.totalGlobalMem / 1048576.0f;
    }

    return memMiBs;
}
