#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hipblas.h>

#include <stdio.h>

#include "cudaUti.cuh"


// basic routine. compute C = AB.
// C, A and B must be consecutive.
// could set the cuda stream.
void gemmcublas(float *C, float *A, float *B, int M, int K, int N, hipblasHandle_t handle, hipStream_t stream) {
    hipblasStatus_t hipblasStatus_t = HIPBLAS_STATUS_SUCCESS;
    hipblasStatus_t = hipblasSetStream(handle, stream);
    CHECKCUBLAS(hipblasStatus_t);

    //void *workspace{ nullptr };
    //CHECKCUDA(hipMallocAsync(&workspace, 1024 * 1024, stream));
    //CHECKCUBLAS(hipblasSetWorkspace(handle, workspace, 1024 * 1024));

    hipError_t cudaStatus;
    float *d_A{nullptr}, *d_B{nullptr}, *d_C{nullptr};
    cudaStatus = hipMallocAsync(&d_A, sizeof(float) * M * K, stream);
    CHECKCUDA(cudaStatus);
    cudaStatus = hipMallocAsync(&d_B, sizeof(float) * K * N, stream);
    CHECKCUDA(cudaStatus);
    cudaStatus = hipMallocAsync(&d_C, sizeof(float) * M * N, stream);
    CHECKCUDA(cudaStatus);

    hipblasStatus_t = hipblasSetMatrixAsync(M, K, sizeof(float), A, M, d_A, M, stream);
    CHECKCUBLAS(hipblasStatus_t);
    hipblasStatus_t = hipblasSetMatrixAsync(K, N, sizeof(float), B, K, d_B, K, stream);
    CHECKCUBLAS(hipblasStatus_t);

    float alpha = 1.0f, beta = 0.0f;

    hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, M, N, K, &alpha, d_A, M, d_B, K, &beta, d_C, M);
    hipblasStatus_t = hipblasGetMatrixAsync(M, N, sizeof(float), d_C, M, C, M, stream);
    CHECKCUBLAS(hipblasStatus_t);

    cudaStatus = hipFreeAsync(d_A, stream);
    CHECKCUDA(cudaStatus);
    cudaStatus = hipFreeAsync(d_B, stream);
    CHECKCUDA(cudaStatus);
    cudaStatus = hipFreeAsync(d_C, stream);
    CHECKCUDA(cudaStatus);
}

// basic routine. compute C = AB.
// C, A and B must be consecutive.
void gemmcublas(float *C, float *A, float *B, int M, int K, int N, hipblasHandle_t handle) {
    float *d_A{nullptr}, *d_B{nullptr}, *d_C{nullptr};
    hipMalloc(&d_A, sizeof(float) * M * K);
    hipMalloc(&d_B, sizeof(float) * K * N);
    hipMalloc(&d_C, sizeof(float) * M * N);

    hipblasSetMatrix(M, K, sizeof(float), A, M, d_A, M);
    hipblasSetMatrix(K, N, sizeof(float), B, K, d_B, K);

    float alpha = 1.0f, beta = 0.0f;

    hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, M, N, K, &alpha, d_A, M, d_B, K, &beta, d_C, M);
    hipblasGetMatrix(N, N, sizeof(float), d_C, M, C, M);

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
}

// get the number of system.
int getdevicecount() {
    int deviceCount = 0;
    CHECKCUDA(hipGetDeviceCount(&deviceCount));
    return deviceCount;
}

// get properties of each GPU: memory.
float getdeviceprop(int dev) {
    float mem;
    hipDeviceProp_t deviceProp;
    CHECKCUDA(hipGetDeviceProperties(&deviceProp, dev));
    mem = deviceProp.totalGlobalMem / 1048576.0f;

    return mem;
}

// block size 32, 2
__global__ void GPU8_mul(float *d_C, int ldC, float *d_A, int ldA, float *d_B, int ldB) {
    __shared__ float bs[16][32];

    float c[16] = {0};
    float a;

    int blockXTimes64 = blockIdx.x * 64;
    int blockYTimes16 = blockIdx.y * 16;
    int tid = threadIdx.y * 32 + threadIdx.x;

    int cNext = blockYTimes16 * ldC + blockXTimes64 + tid;
    int aNext = blockXTimes64 + tid;
    int bNext = (blockYTimes16 + threadIdx.y) * ldB + threadIdx.x;

    d_C += cNext;
    d_A += aNext;
    d_B += bNext;
    float *d_BTmp = d_B;

    int nDiv16 = ldB / 32;

    for (int i = 0; i < nDiv16; ++i) {
        bs[threadIdx.y][threadIdx.x] = d_BTmp[0];
        d_BTmp += 2 * ldB;
        bs[threadIdx.y + 2][threadIdx.x] = d_BTmp[0];
        d_BTmp += 2 * ldB;
        bs[threadIdx.y + 4][threadIdx.x] = d_BTmp[0];
        d_BTmp += 2 * ldB;
        bs[threadIdx.y + 6][threadIdx.x] = d_BTmp[0];
        d_BTmp += 2 * ldB;
        bs[threadIdx.y + 8][threadIdx.x] = d_BTmp[0];
        d_BTmp += 2 * ldB;
        bs[threadIdx.y + 10][threadIdx.x] = d_BTmp[0];
        d_BTmp += 2 * ldB;
        bs[threadIdx.y + 12][threadIdx.x] = d_BTmp[0];
        d_BTmp += 2 * ldB;
        bs[threadIdx.y + 14][threadIdx.x] = d_BTmp[0];
        d_BTmp += 2 * ldB;

        for (int j = 0; j < 32; ++j) {
            a = d_A[0];
            c[0] += a * bs[0][j];
            c[1] += a * bs[1][j];
            c[2] += a * bs[2][j];
            c[3] += a * bs[3][j];
            c[4] += a * bs[4][j];
            c[5] += a * bs[5][j];
            c[6] += a * bs[6][j];
            c[7] += a * bs[7][j];
            c[8] += a * bs[8][j];
            c[9] += a * bs[9][j];
            c[10] += a * bs[10][j];
            c[11] += a * bs[11][j];
            c[12] += a * bs[12][j];
            c[13] += a * bs[13][j];
            c[14] += a * bs[14][j];
            c[15] += a * bs[15][j];

            d_A += ldA;
        }

        d_B += 32;
        d_BTmp = d_B;
    }

    for (int i = 0; i < 16; ++i) {
        d_C[0] = c[i];
        d_C += ldC;
    }
}

// block size 32, 32
__global__ void GPU8_add(float *d_C, int ldC, float *d_A, int ldA, float *d_B, int ldB, int M, int N) {
    int y = 32 * blockIdx.y + threadIdx.y;
    int x = 32 * blockIdx.x + threadIdx.x;
    int idxC = y * ldC + x;
    int idxA = y * ldA + x;
    int idxB = y * ldB + x;
    if (x < M && y < N) {
        d_C[idxC] = d_A[idxA] + d_B[idxB];
    }
}

// block size 32, 32
__global__ void GPU8_sub(float *d_C, int ldC, float *d_A, int ldA, float *d_B, int ldB, int M, int N) {
    int y = 32 * blockIdx.y + threadIdx.y;
    int x = 32 * blockIdx.x + threadIdx.x;
    int idxC = y * ldC + x;
    int idxA = y * ldA + x;
    int idxB = y * ldB + x;
    if (x < M && y < N) {
        d_C[idxC] = d_A[idxA] - d_B[idxB];
    }
}