#include <hip/hip_runtime.h>
#include <hipblas.h>

#include <stdio.h>

#include "cudaUti.cuh"


// basic routine. compute C = AB.
// C, A and B must be consecutive.
// could set the cuda stream.
void gemmcublas(float *C, float *A, float *B, int M, int K, int N, hipblasHandle_t handle, hipStream_t stream) {
    hipblasStatus_t hipblasStatus_t = HIPBLAS_STATUS_SUCCESS;
    hipblasStatus_t = hipblasSetStream(handle, stream);
    CHECKCUBLAS(hipblasStatus_t);

    //void *workspace{ nullptr };
    //CHECKCUDA(hipMallocAsync(&workspace, 1024 * 1024, stream));
    //CHECKCUBLAS(hipblasSetWorkspace(handle, workspace, 1024 * 1024));

    hipError_t cudaStatus;
    float *d_A{nullptr}, *d_B{nullptr}, *d_C{nullptr};
    cudaStatus = hipMallocAsync(&d_A, sizeof(float) * M * K, stream);
    CHECKCUDA(cudaStatus);
    cudaStatus = hipMallocAsync(&d_B, sizeof(float) * K * N, stream);
    CHECKCUDA(cudaStatus);
    cudaStatus = hipMallocAsync(&d_C, sizeof(float) * M * N, stream);
    CHECKCUDA(cudaStatus);

    hipblasStatus_t = hipblasSetMatrixAsync(M, K, sizeof(float), A, M, d_A, M, stream);
    CHECKCUBLAS(hipblasStatus_t);
    hipblasStatus_t = hipblasSetMatrixAsync(K, N, sizeof(float), B, K, d_B, K, stream);
    CHECKCUBLAS(hipblasStatus_t);

    float alpha = 1.0f, beta = 0.0f;

    hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, M, N, K, &alpha, d_A, M, d_B, K, &beta, d_C, M);
    hipblasStatus_t = hipblasGetMatrixAsync(M, N, sizeof(float), d_C, M, C, M, stream);
    CHECKCUBLAS(hipblasStatus_t);

    cudaStatus = hipFreeAsync(d_A, stream);
    CHECKCUDA(cudaStatus);
    cudaStatus = hipFreeAsync(d_B, stream);
    CHECKCUDA(cudaStatus);
    cudaStatus = hipFreeAsync(d_C, stream);
    CHECKCUDA(cudaStatus);
}

// basic routine. compute C = AB.
// C, A and B must be consecutive.
void gemmcublas(float *C, float *A, float *B, int M, int K, int N, hipblasHandle_t handle) {
    float *d_A{nullptr}, *d_B{nullptr}, *d_C{nullptr};
    hipMalloc(&d_A, sizeof(float) * M * K);
    hipMalloc(&d_B, sizeof(float) * K * N);
    hipMalloc(&d_C, sizeof(float) * M * N);

    hipblasSetMatrix(M, K, sizeof(float), A, M, d_A, M);
    hipblasSetMatrix(K, N, sizeof(float), B, K, d_B, K);

    float alpha = 1.0f, beta = 0.0f;

    hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, M, N, K, &alpha, d_A, M, d_B, K, &beta, d_C, M);
    hipblasGetMatrix(N, N, sizeof(float), d_C, M, C, M);

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
}

// get the number of system.
int getdevicecount() {
    int deviceCount = 0;
    CHECKCUDA(hipGetDeviceCount(&deviceCount));
    return deviceCount;
}

// get properties of each GPU: memory.
float getdeviceprop(int dev) {
    float mem;
    hipDeviceProp_t deviceProp;
    CHECKCUDA(hipGetDeviceProperties(&deviceProp, dev));
    mem = deviceProp.totalGlobalMem / 1048576.0f;

    return mem;
}
